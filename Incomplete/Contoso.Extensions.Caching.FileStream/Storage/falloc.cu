#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include <falloc.h>

///////////////////////////////////////////////////////////////////////////////
// STRUCT
// Structures used by device-size code
#pragma region STRUCT

typedef struct __align__(8)
{
	unsigned short magic;		// magic number says we're valid
	unsigned short count;		// number of chunks in sequence
	unsigned short chunkid;		// chunk ID of author
	unsigned short threadid;	// thread ID of author
} fallocChunkHeader;

typedef struct __align__(8)
{
	fallocChunkHeader *chunk;	// chunk reference
	unsigned short chunkid;		// chunk ID of author
	unsigned short threadid;	// thread ID of author
} fallocChunkRef;

typedef struct __align__(8) cuFallocDeviceHeap
{
	void *reserved;
	size_t chunkSize;
	size_t chunksLength;
	size_t chunkRefsLength; // Size of circular buffer (set up by host)
	fallocChunkRef *chunkRefs; // Start of circular buffer (set up by host)
	volatile fallocChunkRef *freeChunkPtr; // Current atomically-incremented non-wrapped offset
	volatile fallocChunkRef *retnChunkPtr; // Current atomically-incremented non-wrapped offset
	char *chunks;
} cuFallocDeviceHeap;

#pragma endregion

///////////////////////////////////////////////////////////////////////////////
// HOST SIDE
// External function definitions for host-side code
#pragma region HOST SIDE

//	cudaFallocSetDefaultHeap
extern "C" hipError_t cudaFallocSetDefaultHeap(cudaDeviceFallocHeap &heap)
{
	return hipMemcpyToSymbol(HIP_SYMBOL(_defaultDeviceHeap), &heap.deviceHeap, sizeof(cuFallocDeviceHeap *));
}

//  cudaDeviceFallocCreate
//
//  Takes a buffer length to allocate, creates the memory on the device and
//  returns a pointer to it for when a kernel is called. It's up to the caller
//  to free it.
static __forceinline__ void writeChunkRefHost(fallocChunkRef *ref, fallocChunkHeader *chunk) { ref->chunk = chunk; ref->chunkid = 0; ref->threadid = 0; }
extern "C" cudaDeviceFallocHeap cudaDeviceFallocHeapCreate(size_t chunkSize, size_t length, hipError_t *error, void *reserved)
{
	hipError_t localError; if (!error) error = &localError;
	cudaDeviceFallocHeap heap; memset(&heap, 0, sizeof(cudaDeviceFallocHeap));
	// fix up chunkSize to include fallocChunkHeader
	chunkSize = (chunkSize + sizeof(fallocChunkHeader) + 15) & ~15;
	// fix up length to be a multiple of chunkSize
	if (!length || length % chunkSize)
		length += chunkSize - (length % chunkSize);
	size_t chunksLength = length;
	size_t chunks = (size_t)(chunksLength / chunkSize);
	if (!chunks)
		return heap;
	// fix up length to include cuFallocDeviceHeap + freechunks
	unsigned int chunkRefsLength = (unsigned int)(chunks * sizeof(fallocChunkRef));
	length = (length + chunkRefsLength + sizeof(cuFallocDeviceHeap) + 15) & ~15;
	// allocate a heap on the device and zero it
	cuFallocDeviceHeap *deviceHeap;
	if ((*error = hipMalloc((void **)&deviceHeap, length)) != hipSuccess || (*error = hipMemset(deviceHeap, 0, length)) != hipSuccess)
		return heap;
	// transfer to heap
	cuFallocDeviceHeap hostDeviceHeap;
	hostDeviceHeap.reserved = reserved;
	hostDeviceHeap.chunkSize = chunkSize;
	hostDeviceHeap.chunksLength = chunksLength;
	hostDeviceHeap.chunkRefsLength = chunkRefsLength;
	hostDeviceHeap.chunkRefs = (fallocChunkRef *)((char *)deviceHeap + sizeof(cuFallocDeviceHeap));
	hostDeviceHeap.freeChunkPtr = hostDeviceHeap.retnChunkPtr = (volatile fallocChunkRef *)hostDeviceHeap.chunkRefs;
	hostDeviceHeap.chunks = (char *)hostDeviceHeap.chunkRefs + chunkRefsLength;
	if ((*error = hipMemcpy(deviceHeap, &hostDeviceHeap, sizeof(cuFallocDeviceHeap), hipMemcpyHostToDevice)) != hipSuccess)
		return heap;
	// initial chunkrefs
	char *chunk = hostDeviceHeap.chunks;
	fallocChunkRef *hostChunkRefs = new fallocChunkRef[chunks];
	unsigned int i;
	fallocChunkRef *r;
	for (i = 0, r = hostChunkRefs; i < chunks; i++, r++, chunk += chunkSize)
		writeChunkRefHost(r, (fallocChunkHeader *)chunk);
	// transfer to heap
	*error = hipMemcpy(hostDeviceHeap.chunkRefs, hostChunkRefs, sizeof(fallocChunkRef) * chunks, hipMemcpyHostToDevice);
	delete hostChunkRefs;
	if (*error != hipSuccess)
		return heap;
	// return the heap
	heap.reserved = reserved;
	heap.deviceHeap = deviceHeap;
	heap.chunkSize = chunkSize;
	heap.chunksLength = chunksLength;
	heap.length = length;
	return heap;
}

//  cudaDeviceFallocHeapDestroy
//
//  Frees up the memory which we allocated
extern "C" hipError_t cudaDeviceFallocHeapDestroy(cudaDeviceFallocHeap &heap)
{
	if (!heap.deviceHeap)
		return hipSuccess;
	hipError_t error = hipFree(heap.deviceHeap); heap.deviceHeap = nullptr;
	return error;
}

#pragma endregion

///////////////////////////////////////////////////////////////////////////////
// DEVICE SIDE :: HEAP
// Heap function definitions for device-side code
#pragma region DEVICE SIDE :: HEAP

#if defined(__CUDA_ARCH__)
#define panic(fmt) { printf(fmt"\n"); asm("trap;"); }
#else
#define panic(fmt) { printf(fmt"\n"); exit(1); }
#endif  /* __CUDA_ARCH__ */

__constant__ cuFallocDeviceHeap *_defaultDeviceHeap;

#define FALLOC_MAGIC (unsigned short)0x3412 // All our headers are prefixed with a magic number so we know they're ours

static __device__ __forceinline__ void writeChunkRef(fallocChunkRef *ref, fallocChunkHeader *chunk)
{
	ref->chunk = chunk;
	ref->chunkid = gridDim.x*blockIdx.y + blockIdx.x;
	ref->threadid = blockDim.x*blockDim.y*threadIdx.z + blockDim.x*threadIdx.y + threadIdx.x;
}

static __device__ __forceinline__ void writeChunkHeader(fallocChunkHeader *hdr, unsigned short count)
{
	fallocChunkHeader header;
	header.magic = FALLOC_MAGIC;
	header.count = count;
	header.chunkid = gridDim.x*blockIdx.y + blockIdx.x;
	header.threadid = blockDim.x*blockDim.y*threadIdx.z + blockDim.x*threadIdx.y + threadIdx.x;
	*hdr = header;
}

extern "C" __device__ void *fallocGetChunk(cuFallocDeviceHeap *heap)
{
	if (!heap) heap = _defaultDeviceHeap;
	// advance circular buffer
	fallocChunkRef *chunkRefs = heap->chunkRefs;
	size_t offset = atomicAdd((unsigned int *)&heap->freeChunkPtr, sizeof(fallocChunkRef)) - (size_t)chunkRefs;
	offset %= heap->chunkRefsLength;
	fallocChunkRef *chunkRef = (fallocChunkRef *)((char *)chunkRefs + offset);
	fallocChunkHeader *chunk = chunkRef->chunk;
	writeChunkHeader(chunk, 1);
	chunkRef->chunk = nullptr;
	return (void *)((char *)chunk + sizeof(fallocChunkHeader));
}

extern "C" __device__ void fallocFreeChunk(void *obj, cuFallocDeviceHeap *heap)
{
	if (!heap) heap = _defaultDeviceHeap;
	fallocChunkHeader *chunk = (fallocChunkHeader *)((char *)obj - sizeof(fallocChunkHeader));
	if (chunk->magic != FALLOC_MAGIC || chunk->count > 1) panic("bad magic"); // bad magic or not a singular chunk
	// advance circular buffer
	fallocChunkRef *chunkRefs = heap->chunkRefs;
	size_t offset = atomicAdd((unsigned int *)&heap->retnChunkPtr, sizeof(fallocChunkRef)) - (size_t)chunkRefs;
	offset %= heap->chunkRefsLength;
	writeChunkRef((fallocChunkRef *)((char *)chunkRefs + offset), chunk);
	chunk->magic = 0;
}

#if MULTIBLOCK
/*
extern "C" __device__ inline void *fallocGetChunks(fallocHeap *heap, size_t length, size_t *allocLength = nullptr)
{
if (threadIdx.x || threadIdx.y || threadIdx.z) panic("");
size_t chunkSize = heap->chunkSize;
// fix up length to be a multiple of chunkSize
if (length % chunkSize)
length += chunkSize - (length % chunkSize);
// set length, if requested
if (allocLength)
*allocLength = length - sizeof(fallocChunkHeader);
size_t chunks = (size_t)(length / chunkSize);
if (chunks > heap->chunks) panic("");
// single, equals: fallocGetChunk
if (chunks == 1)
return fallocGetChunk(heap);
// multiple, find a contiguous chuck
size_t index = chunks;
volatile fallocChunkHeader* chunk;
volatile fallocChunkHeader* endChunk = (fallocChunkHeader*)((__int8*)heap + sizeof(fallocHeap) + (chunkSize * heap->chunks));
{ // critical
for (chunk = (fallocChunkHeader*)((__int8*)heap + sizeof(fallocHeap)); index && chunk < endChunk; chunk = (fallocChunkHeader*)((__int8*)chunk + (chunkSize * chunk->count)))
{
if (chunk->magic != FALLOC_MAGIC) panic("bad magic");
index = (chunk->next ? index - 1 : chunks);
}
if (index)
return nullptr;
// found chuck, remove from chunkRefs
endChunk = chunk;
chunk = (fallocChunkHeader*)((__int8*)chunk - (chunkSize * chunks));
for (volatile fallocChunkHeader* chunk2 = heap->chunkRefs; chunk2; chunk2 = chunk2->next)
if (chunk2 >= chunk && chunk2 <= endChunk)
chunk2->next = (chunk2->next ? chunk2->next->next : nullptr);
chunk->count = chunks;
chunk->next = nullptr;
}
return (void*)((__int8*)chunk + sizeof(fallocChunkHeader));
}

extern "C" __device__ inline void fallocFreeChunks(fallocHeap *heap, void *obj)
{
volatile fallocChunkHeader* chunk = (fallocChunkHeader*)((__int8*)obj - sizeof(fallocChunkHeader));
if (chunk->magic != FALLOC_MAGIC) panic("bad magic");
size_t chunks = chunk->count;
// single, equals: fallocFreeChunk
if (chunks == 1)
{
{ // critical
chunk->next = heap->chunkRefs;
heap->chunkRefs = chunk;
}
return;
}
// retag chunks
size_t chunkSize = heap->chunkSize;
chunk->count = 1;
while (chunks-- > 1)
{
chunk = chunk->next = (fallocChunkHeader*)((__int8*)chunk + sizeof(fallocChunkHeader) + chunkSize);
chunk->magic = FALLOC_MAGIC;
chunk->count = 1;
chunk->reserved = nullptr;
}
{ // critical
chunk->next = heap->chunkRefs;
heap->chunkRefs = chunk;
}
}
*/
#endif

#pragma endregion

///////////////////////////////////////////////////////////////////////////////
// DEVICE SIDE :: CONTEXT
// Context function definitions for device-side code
#pragma region DEVICE SIDE :: CONTEXT

const static int FALLOCNODE_SLACK = 0x10;
#define FALLOCNODE_MAGIC (unsigned short)0x7856 // All our headers are prefixed with a magic number so we know they're ours
#define FALLOCCTX_MAGIC (unsigned short)0xCC56 // All our headers are prefixed with a magic number so we know they're ours

typedef struct __align__(8) cuFallocNode
{
	struct cuFallocNode *next;
	struct cuFallocNode *nextAvailable;
	unsigned short freeOffset;
	unsigned short magic;
} fallocNode;

typedef struct __align__(8) cuFallocCtx
{
	fallocNode node;
	fallocNode *nodes;
	fallocNode *availableNodes;
	cuFallocDeviceHeap *heap;
	size_t chunkSize;
	unsigned short magic;
} cuFallocCtx;

extern "C" __device__ cuFallocCtx *fallocCreateCtx(cuFallocDeviceHeap *heap)
{
	if (!heap) heap = _defaultDeviceHeap;
	size_t chunkSize = heap->chunkSize;
	if (sizeof(cuFallocCtx) > chunkSize) panic("large chucksize");
	cuFallocCtx *ctx = (cuFallocCtx *)fallocGetChunk(heap);
	if (!ctx)
		return nullptr;
	ctx->node.magic = FALLOCNODE_MAGIC;
	ctx->node.next = nullptr;
	ctx->node.nextAvailable = nullptr;
	unsigned short freeOffset = ctx->node.freeOffset = sizeof(cuFallocCtx);
	ctx->nodes = (fallocNode *)ctx;
	ctx->availableNodes = (fallocNode *)ctx;
	ctx->heap = heap;
	ctx->chunkSize = heap->chunkSize;
	ctx->magic = FALLOCCTX_MAGIC;
	// close node
	if (freeOffset + FALLOCNODE_SLACK > chunkSize)
		ctx->availableNodes = nullptr;
	return ctx;
}

extern "C" __device__ void fallocDisposeCtx(cuFallocCtx *ctx)
{
	cuFallocDeviceHeap *heap = ctx->heap;
	for (fallocNode *node = ctx->nodes; node; node = node->next)
		fallocFreeChunk(node, heap);
}

extern "C" __device__ void *falloc(cuFallocCtx *ctx, unsigned short bytes, bool alloc)
{
	if (bytes > (ctx->chunkSize - sizeof(cuFallocCtx))) panic("size");
	// find or add available node
	fallocNode *node;
	unsigned short freeOffset;
	unsigned char hasFreeSpace;
	fallocNode *lastNode;
	for (lastNode = (fallocNode *)ctx, node = ctx->availableNodes; node; lastNode = node, node = (alloc ? node->nextAvailable : node->next))
		if (hasFreeSpace = ((freeOffset = node->freeOffset + bytes) <= ctx->chunkSize))
			break;
	if (!node || !hasFreeSpace) {
		// add node
		node = (fallocNode *)fallocGetChunk(ctx->heap);
		if (!node) panic("alloc");
		node->magic = FALLOCNODE_MAGIC;
		node->next = ctx->nodes; ctx->nodes = node;
		node->nextAvailable = (alloc ? ctx->availableNodes : nullptr); ctx->availableNodes = node;
		freeOffset = node->freeOffset = sizeof(fallocNode); 
		freeOffset += bytes;
	}
	//
	void *obj = (char *)node + node->freeOffset;
	node->freeOffset = freeOffset;
	// close node
	if (alloc && (freeOffset + FALLOCNODE_SLACK > ctx->chunkSize)) {
		if (lastNode == (fallocNode *)ctx)
			ctx->availableNodes = node->nextAvailable;
		else
			lastNode->nextAvailable = node->nextAvailable;
		node->nextAvailable = nullptr;
	}
	return obj;
}

extern "C" __device__ void *fallocRetract(cuFallocCtx *ctx, unsigned short bytes)
{
	fallocNode *node = ctx->availableNodes;
	int freeOffset = (int)node->freeOffset - bytes;
	// multi node, retract node
	if (node != &ctx->node && freeOffset < sizeof(fallocNode)) {
		node->freeOffset = sizeof(fallocNode);
		// search for previous node
		fallocNode *lastNode;
		for (lastNode = (fallocNode *)ctx, node = ctx->nodes; node; lastNode = node, node = node->next)
			if (node == ctx->availableNodes)
				break;
		node = ctx->availableNodes = lastNode;
		freeOffset = (int)node->freeOffset - bytes;
	}
	// first node && !overflow
	if (node == &ctx->node && freeOffset < sizeof(cuFallocCtx)) panic("node");
	node->freeOffset = (unsigned short)freeOffset;
	return (char *)node + freeOffset;
}

extern "C" __device__ void fallocMark(cuFallocCtx *ctx, void *&mark, unsigned short &mark2) { mark = ctx->availableNodes; mark2 = ctx->availableNodes->freeOffset; }
extern "C" __device__ bool fallocAtMark(cuFallocCtx *ctx, void *mark, unsigned short mark2) { return (mark == ctx->availableNodes && mark2 == ctx->availableNodes->freeOffset); }

#pragma endregion